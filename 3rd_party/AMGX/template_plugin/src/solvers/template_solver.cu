#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <solvers/template_solver.h>

namespace amgx
{
namespace template_plugin
{

template<class T_Config>
TemplateSolver<T_Config>::TemplateSolver( AMG_Config &cfg, const std::string &cfg_scope) : Solver<T_Config>( cfg, cfg_scope) {}

template<class T_Config>
void
TemplateSolver<T_Config>::solver_setup(bool reuse_matrix_structure)
{
    // Allocations and initalizations should be done in setup
}

template<class T_Config>
bool
TemplateSolver<T_Config>::solve_iteration( VVector &b, VVector &x, bool xIsZero )
{
    // One iterations of this linear solver. Modify x somehow to reduce residual norm | b - A*x |. A and b should not be modified here.
    return this->converged( b, x );
};

/****************************************
 * Explict instantiations
 ***************************************/
#define AMGX_CASE_LINE(CASE) template class TemplateSolver<TemplateMode<CASE>::Type>;
AMGX_FORALL_BUILDS(AMGX_CASE_LINE)
#undef AMGX_CASE_LINE
} // namespace template_plugin
} // namespace amgx
