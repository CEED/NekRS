
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define p_blockSize 256

extern "C" __global__ void packBuf_half(
  const int Nscatter,
  const int Nentries,
  const int * __restrict__ scatterStarts,
  const int * __restrict__ scatterIds,
  const float * __restrict__ q,
  half * __restrict__ scatterq
)
{
  int tile = p_blockSize * blockIdx.x;
  {
    int s = tile + threadIdx.x;
    if (s < Nscatter * Nentries) {
      const float qs = q[s];
      const int sid = s % Nscatter;
      const int k = s / Nscatter;
      const int start = scatterStarts[sid];
      const int end = scatterStarts[sid + 1];
      for (int n = start; n < end; ++n) {
        const int id = scatterIds[n];
        scatterq[id * Nentries + k] = __float2half(qs);
      }
    }
  }
}

extern "C" __global__ void unpackBuf_halfAdd(const int Ngather,
                                             const int Nentries,
                                             const int * __restrict__ gatherStarts,
                                             const int * __restrict__ gatherIds,
                                             const half * __restrict__ q,
                                             float * __restrict__ gatherq) {
  {
    int tile = p_blockSize * blockIdx.x;
    {
      int g = tile + threadIdx.x;
      if (g < Ngather * Nentries) {
        const int gid = g % Ngather;
        const int k = g / Ngather;
        const int start = gatherStarts[gid];
        const int end = gatherStarts[gid + 1];
        float gq = 0.00000000e+00f;
        for (int n = start; n < end; ++n) {
          const int id = gatherIds[n];
          gq += __half2float(q[id * Nentries + k]);
        }

        //contiguously packed
        gatherq[g] += gq;
      }
    }
  }
}
